#include "hip/hip_runtime.h"
#include <string>
#include <iomanip>
#include <sstream>
#include <utility>
#include <cmath>
#include <thread>
#include <vector>
#include <atomic>
#include <memory>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime_api.h>
#include "util.hpp"
#include "protocol.hpp"

#define cudaAssert(ans) { gpuAssert_impl((ans), __FILE__, __LINE__); }
inline void cudaCheck(){
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
}
inline void gpuAssert_impl(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// -+-+-+-+-+-+-+-+-+-+- //
//       Main Loop       //
// -+-+-+-+-+-+-+-+-+-+- //

__global__ void clearGpuMemory(unsigned char* ptr, int size, int parallel){
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int lb =  idx    * parallel;
	int ub = (idx+1) * parallel;
	ub = ub < size ? ub : size;
	for(int i = lb; i < ub; ++i)
		ptr[i] = 0;
}

__device__ void invoke_render_impl(unsigned char* img, const Status status, int offset){
	const int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;
	const int y = idx / status.width;
	const int x = idx % status.width;
	if(y < status.height){
		unsigned char* ptr = img + (y * status.width + x) * 4;
		RGBA col = renderer_gpu::render(img, status, x, y);
		ptr[0] = col.b;
		ptr[1] = col.g;
		ptr[2] = col.r;
		ptr[3] = col.a;
	}
}

__global__ void invoke_render(unsigned char* img, const Status status, int offset){
	invoke_render_impl(img, status, offset);
}

int main(int argc, char *argv[]){
	const float fps_      = (argc > 1 ? std::stof(argv[1]) :   30);  // デフォルト: 30 fps
	const int   width_    = (argc > 2 ? std::stoi(argv[2]) : 1920);  // デフォルト: 1920 px
	const int   height_   = (argc > 3 ? std::stoi(argv[3]) : 1080);  // デフォルト: 1080 px

	Status status{ 0, fps_, 0, 0, height_, width_ };
	renderer_cpu::init(status);

	std::cout << "fps: "       << status.fps      << std::endl;
	std::cout << "duration: "  << status.duration << std::endl;
	std::cout << "width: "     << status.width    << std::endl;
	std::cout << "height: "    << status.height   << std::endl;

	std::atomic_int done_frame_cnt{0};
	int total_frame_cnt = status.fps * status.duration;

	// GPUメモリ確保
	unsigned char* device_img_raw;
	cudaAssert(hipMalloc(reinterpret_cast<void**>(&device_img_raw), status.width*status.height*4*sizeof(unsigned char)));
	std::shared_ptr<unsigned char> device_img(device_img_raw, hipFree);
	clearGpuMemory<<<status.width/4+1,status.height/4+1>>>(device_img.get(), status.width*status.height*4, 64);
	cudaCheck();
	
	// CPUメモリ確保
	cv::Mat img = cv::Mat::zeros(cv::Size(status.width, status.height), CV_MAKE_TYPE(CV_8U, 4));
	
	// メインループ
	for(int frame = 0; frame < total_frame_cnt; ++frame){
		float time = float(frame) / status.fps;

		Status current_status = status;
		current_status.frame = frame;
		current_status.time  = time;
		for(int i=0; i*1024*1024 < status.height * status.width; ++i)
			invoke_render<<<1024,1024>>>(device_img_raw, current_status, i*1024*1024);
		cudaCheck();

		cudaAssert(hipMemcpy(img.data, device_img.get(), status.width*status.height*4, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();

		std::ostringstream file_name;
		file_name << "png/out_" << zero_ume(frame) << ".png";
		cv::imwrite(file_name.str(), img);

		progress_bar(frame, total_frame_cnt);
	}
	std::cerr << std::endl;
}


// -+-+-+-+-+-+-+-+-+-+- //
//        Include        //
// -+-+-+-+-+-+-+-+-+-+- //

#include "main.hpp"
